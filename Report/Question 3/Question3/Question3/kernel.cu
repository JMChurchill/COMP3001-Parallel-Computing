#include "hip/hip_runtime.h"
﻿/*
------------------DR VASILIOS KELEFOURAS-----------------------------------------------------
------------------COMP3001 ------------------------------------------------------------------
------------------PARALLEL PROGAMMING MODULE-------------------------------------------------
------------------UNIVERSITY OF PLYMOUTH, SCHOOL OF ENGINEERING, COMPUTING AND MATHEMATICS---
*/

// Optimised Elapsed time: 273.22 msecs
// Optimised Flops value: 31.44 Gflops


#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <omp.h>

//#define BILLION 1000000000
#define TIMES_TO_RUN 1

//#define ARITHMETICAL_OPS 2 * N * N * N * N
#define ARITHMETICAL_OPS 8589934592

#define N 256 //input size
#define TILE 8

__declspec(align(64)) float test[N][N][N], sum[N][N][N], A[N][N][N], C[N][N];

__device__ float device_sum[N][N][N], device_A[N][N][N], device_C[N][N], device_T[N][N][N]; //allocate the device arrays statically (global GPU memory)

void init();
void default();
int Compare();
inline unsigned short int equal(float const a, float const b);


#define EPSILON 0.00001

#define MAX_NUMBER_OF_BLOCKS_PER_DIM 65535 //max number of blocks that our GPU can handle (for one dimension only)


//template<typename T> __global__ void diotgen_ver1(T * test, T * A, T * C) {
 __global__ void diotgen_ver1() {
	 float temp = 0.0;

	 int r = blockIdx.x * blockDim.x + threadIdx.x; //2d grid, 2d blocks
	 int q = blockIdx.y * blockDim.y + threadIdx.y;
	 int p = blockIdx.z * blockDim.z + threadIdx.z;

	 if (r < N && q < N && p < N) {

		 for (int s = 0; s < N; s++) {
			 temp += device_A[r][q][s] * device_C[s][p];
		 }
		 device_sum[r][q][p] = temp;
	 }

	 //original
	 /*for (int r = 0; r < N; r++)
		 for (int q = 0; q < N; q++)
			 for (int p = 0; p < N; p++)
				 for (int s = 0; s < N; s++)
					 device_sum[r][q][p] = device_sum[r][q][p] + device_A[r][q][s] * device_C[s][p];*/
}


 //__global__ void diotgen_ver1T() {
	// float temp = 0.0;

	// int r = blockIdx.x * blockDim.x + threadIdx.x; //2d grid, 2d blocks
	// int q = blockIdx.y * blockDim.y + threadIdx.y;
	// int p = blockIdx.z * blockDim.z + threadIdx.z;
	// //if(r == 0 && q == 0)
	// //printf("\n%d, %d, %d", r, q,p);

	// //for (int r = 0; r < N; r++)
	//	 //for (int q = 0; q < N; q++)
	// if (r < N && q < N && p < N) {

	//	 //for (int p = 0; p < N; p++)
	//	 for (int s = 0; s < N; s++) {
	//		 //if (r == 0 && q == 0)
	//			// printf("\n%d, %d, %d, %d", r, q,p,s);
	//		 temp += device_A[r][q][s] * device_C[s][p];
	//	 }
	//	 device_sum[r][q][p] = temp;
	// }




	// //original
	// /*for (int r = 0; r < N; r++)
	//	 for (int q = 0; q < N; q++)
	//		 for (int p = 0; p < N; p++)
	//			 for (int s = 0; s < N; s++)
	//				 device_sum[r][q][p] = device_sum[r][q][p] + device_A[r][q][s] * device_C[s][p];*/
 //}

 __global__ void unoptimiseddiotgen_ver1() {
	//original
	for (int r = 0; r < N; r++)
		for (int q = 0; q < N; q++)
			for (int p = 0; p < N; p++)
				for (int s = 0; s < N; s++)
					device_sum[r][q][p] = device_sum[r][q][p] + device_A[r][q][s] * device_C[s][p];
 }



int main()
{
	hipError_t cudaStatus;
	double my_flops;

	//------create the cuda timers------
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsed_time;

	int devId = 0;
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devId);
	printf("\n Device: %s \n", prop.name);

	init(); //initialize host arrays

	/* Copy the Test array from the HOST memory to the DEVICE memory */
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(device_T), test, N * N * N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("\ncudaMemcpy failed!");
		return -1;
	}

	/* Copy the A array from the HOST memory to the DEVICE memory */
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(device_A), A, N * N * N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("\ncudaMemcpy failed!");
		return -1;
	}

	/* Copy the C array from the HOST memory to the DEVICE memory */
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(device_C), C, N * N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("\ncudaMemcpy failed!");
		return -1;
	}

	hipEventRecord(start, 0); //get timer value

	clock_t start_1, end_1; //ignore this for  now
	start_1 = clock();

	//dim3 dimBlock(1, 1, 1);
	//dim3 dimGrid(1, 1, 1);

	dim3 dimBlock(TILE, TILE, TILE);
	dim3 dimGrid(N/TILE, N/TILE, N/TILE);

	//diotgen_ver1<float> << <dimGrid, dimBlock >> > (d_T, d_A, d_C);
	for (int it = 0; it < TIMES_TO_RUN; it++)
	{
		//unoptimiseddiotgen_ver1 << <dimGrid, dimBlock >> > ();
		diotgen_ver1 << <dimGrid, dimBlock >> > ();
	}
	//default();

	end_1 = clock(); //end the timer
	printf("%ld", ARITHMETICAL_OPS);
	hipEventRecord(stop, 0);  //get timer value
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("\nElapsed time in msecs = %f", elapsed_time);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	double flops = (double)((double)ARITHMETICAL_OPS) / (elapsed_time / TIMES_TO_RUN);
	printf("\nGflops achieved %f ", flops / 1000000);

	/* Copy back the result from the DEVICE memory to the HOST memory */
	cudaStatus = hipMemcpyFromSymbol(sum, HIP_SYMBOL(device_sum), N * N * N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("\ncudaMemcpy failed!");
		return -1;
	}

	//do not forget to print the flops value achieved

	/*  Handling function of the CUDA runtime application programming interface.
	*   Returns the last error from a runtime call.
	*/
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(error));
	}



	if (Compare() != 0)
		printf("\n---------WRONG OUTPUT---------------\n");
	else
		printf("\n---------OUTPUT IS OK---------------\n");


	/* Destroy all allocations and reset all state on the current device in the current process */
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		printf("\ncuda Reset failed!");
		return -1;
	}

	return 0;
}


void init() {

	float e = 0.12, p = 0.72;
	unsigned int i, j, k;

	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			C[i][j] = (j % 9) + p;
		}
	}

	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			for (k = 0; k < N; k++) {
				sum[i][j][k] = 0.0;
				test[i][j][k] = 0.0;
				A[i][j][k] = (((i + j) % 99) + e);
			}
		}
	}


}

//this is the routine that you will parallelize 
void default() {

	for (int r = 0; r < N; r++)
		for (int q = 0; q < N; q++)
			for (int p = 0; p < N; p++)
				for (int s = 0; s < N; s++)
					test[r][q][p] = test[r][q][p] + A[r][q][s] * C[s][p];


}


unsigned short int equal(float const a, float const b) {
	float temp = a - b;
	//printf("\n %f  %f", a, b);
	if (fabs(temp / b) < EPSILON)
		return 0; //success
	else
		return 1;
}


int Compare() {


	for (int r = 0; r < N; r++)
		for (int q = 0; q < N; q++)
			for (int p = 0; p < N; p++)
				for (int s = 0; s < N; s++)
					test[r][q][p] = test[r][q][p] + A[r][q][s] * C[s][p];


	for (int r = 0; r < N; r++)
		for (int q = 0; q < N; q++)
			for (int p = 0; p < N; p++)
				if (equal(sum[r][q][p], test[r][q][p]) == 1) {
					printf("\n wrong at (%d,%d,%d)", r, q, p);
					return -1;
				}
	return 0;
}