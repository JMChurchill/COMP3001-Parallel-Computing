#include "hip/hip_runtime.h"
﻿/*
------------------DR VASILIOS KELEFOURAS-----------------------------------------------------
------------------COMP3001 ------------------------------------------------------------------
------------------PARALLEL PROGAMMING MODULE-------------------------------------------------
------------------UNIVERSITY OF PLYMOUTH, SCHOOL OF ENGINEERING, COMPUTING AND MATHEMATICS---
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <omp.h>

#define TIMES_TO_RUN 1 //how many times the function will run

#define N 128 //input size - USE POWER OF 2 ONLY
//#define CHECK_OUTPUT   //if do not want to validate the results comment this
//#define ARITHMETICAL_OPS 5*N*N*N
//#define ARITHMETICAL_OPS 344100945960//4098
//#define ARITHMETICAL_OPS 42949672960//2048
//#define ARITHMETICAL_OPS 5368709120//1024
//#define ARITHMETICAL_OPS 671088640//512
//#define ARITHMETICAL_OPS 83886080//256
#define ARITHMETICAL_OPS 10485760//128
//#define ARITHMETICAL_OPS 1310720//64


__declspec(align(64)) float C[N * N], test[N * N], A[N * N], B[N * N]; //square matrixes are considered only, stored as 1d arrays

void MMM_init();
void MMM_default();
int Compare_MMM();
inline unsigned short int equal(float const a, float const b);


#define EPSILON 0.00001

#define MAX_NUMBER_OF_BLOCKS_PER_DIM 65535 //max number of blocks that our GPU can handle (for one dimension only)



__global__ void mmm_ver1(float* C, float* A, float* B) {
	//#Implementation #3

	//use dim3 dimBlock(16, 16, 1);
	//use dim3 dimGrid(N/16, N/16, 1);

	float tmp = 0.0;

	int i = blockIdx.x * blockDim.x + threadIdx.x; //i loop has been parallelized
	int j = blockIdx.y * blockDim.y + threadIdx.y; //j loop has been parallelized

	for (int k = 0; k < N; k++) {
		tmp += A[N * i + k] * B[N * k + j];
	}

	C[N * i + j] = tmp;
}

//implementation #4
//use dim3 dimBlock(16, 16, 1);
//use dim3 dimGrid(N/16, N/16, 1);

__global__ void mmm_tiled(float* C, float* A, float* B) {
	__shared__ float aa[16][16];
	__shared__ float bb[16][16];

	float tmp = 0.0;
	int k, m;

	int row_A = 16 * blockIdx.y + threadIdx.y;
	int col_B = blockIdx.x * 16 + threadIdx.x;

	for (m = 0; m < N / 16; m++) {
		aa[threadIdx.y][threadIdx.x] = A[N * (row_A)+(m * 16 + threadIdx.x)];
		bb[threadIdx.y][threadIdx.x] = B[N * (m * 16 + threadIdx.y) + (col_B)];

		__syncthreads();

		for (k = 0; k < 16; k++) {
			tmp += aa[threadIdx.y][k] * bb[k][threadIdx.x];
		}

		__syncthreads();
	}

	C[N * row_A + col_B] = tmp;
}



int main()
{
	hipError_t cudaStatus;

	//------create the cuda timers------
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsed_time;

	MMM_init(); //initialize host arrays

	float* C_d, * A_d, * B_d; //pointers to device arrays

	//---------------------------create GPU arrays------------------------------------------
	cudaStatus = hipMalloc((void**)&C_d, N * N * sizeof(float));//allocate memory dynamically
	if (cudaStatus != hipSuccess) {//if the GPU memory asked is not available
		printf("\nCudaMalloc failed");
		hipFree(C_d);
		return -1;//returns unsuccessfully
	}

	cudaStatus = hipMalloc((void**)&A_d, N * N * sizeof(float));//allocate memory dynamically
	if (cudaStatus != hipSuccess) {//if the GPU memory asked is not available
		printf("\nCudaMalloc failed");
		hipFree(C_d); hipFree(A_d);
		return -1;//returns unsuccessfully
	}

	cudaStatus = hipMalloc((void**)&B_d, N * N * sizeof(float));//allocate memory dynamically
	if (cudaStatus != hipSuccess) {//if the GPU memory asked is not available
		printf("\nCudaMalloc failed");
		hipFree(C_d); hipFree(A_d); hipFree(B_d);
		return -1;//returns unsuccessfully
	}



	//--------------------copy arrays from host to device------------------------
	cudaStatus = hipMemcpy(A_d, A, N * N * sizeof(float), hipMemcpyHostToDevice); //copy array from host to GPU
	if (cudaStatus != hipSuccess) {//if cuda copy fails
		printf("\ncuda copy failed");
		hipFree(C_d); hipFree(A_d); hipFree(B_d);
		return -1;//returns unsuccessfully
	}

	cudaStatus = hipMemcpy(B_d, B, N * N * sizeof(float), hipMemcpyHostToDevice); //copy array from host to GPU
	if (cudaStatus != hipSuccess) {//if cuda copy fails
		hipFree(C_d); hipFree(A_d); hipFree(B_d);
		printf("\ncuda copy failed");
		return -1;//returns unsuccessfully
	}


	hipEventRecord(start, 0); //get timer value

	for (int it = 0; it < TIMES_TO_RUN; it++) {

		//dim3 dimBlock(1, 1, 1);
		//dim3 dimGrid(1, 1, 1);

		dim3 dimBlock(16, 16, 1);
		dim3 dimGrid(N / 16, N / 16, 1);
		//mmm_ver1 << <dimGrid, dimBlock >> > (C_d, A_d, B_d);
		mmm_tiled << <dimGrid, dimBlock >> > (C_d, A_d, B_d);


	}


	hipEventRecord(stop, 0);  //get timer value
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("\nElapsed time in msecs = %f", elapsed_time);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	double flops = (double)((double)ARITHMETICAL_OPS) / (elapsed_time / TIMES_TO_RUN);
	printf("\nGflops achieved %f ", flops / 1000000);

	/*  Handling function of the CUDA runtime application programming interface.
	*   Returns the last error from a runtime call.
	*/
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(error));
	}


	cudaStatus = hipMemcpy(C, C_d, N * N * sizeof(float), hipMemcpyDeviceToHost); //copy array from GPU back to CPU
	if (cudaStatus != hipSuccess) {//if cuda copy fails
		printf("\ncuda copy failed");
		hipFree(C_d); hipFree(A_d); hipFree(B_d);
		return -1;//returns unsuccessfully
	}

	//MMM_default();

#ifdef CHECK_OUTPUT
	if (Compare_MMM() != 0)
		printf("\n---------WRONG OUTPUT---------------\n");
	else
		printf("\n---------OUTPUT IS OK---------------\n");
#endif

	/* Destroy all allocations and reset all state on the current device in the current process */
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		printf("\ncuda Reset failed!");
		return -1;
	}

	return 0;
}


void MMM_init() {

	float e = 0.1234, p = 0.7264, r = 0.11;

	//MMM
	for (unsigned int i = 0; i < N; i++) { //printf("\n");
		for (unsigned int j = 0; j < N; j++) {
			C[N * i + j] = 0.0;
			test[N * i + j] = 0.0;
			A[N * i + j] = (j % 9) + p; //printf(" %3.1f",A[i][j]);
			B[N * i + j] = (j % 7) - p; //printf(" %3.1f",B[i][j]);
		}
	}


}


void MMM_default() {

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			for (int k = 0; k < N; k++)
				C[N * i + j] += A[N * i + k] * B[N * k + j];


}


unsigned short int equal(float const a, float const b) {
	float temp = a - b;
	//printf("\n %f  %f", a, b);
	if (fabs(temp / b) < EPSILON)
		return 0; //success
	else
		return 1;
}

int Compare_MMM() {

	float tmp;
	int i, j, k;

	//optimize the following, otherwise it takes too long...however, to allow VS to use the \pragmas you must go
	//in project  properties and enable that (look at the lab session document for more info)
#pragma omp parallel
	{
#pragma omp for private(i, j, k, tmp)
		for (i = 0; i < N; i++) {
			for (j = 0; j < N; j++) {
				tmp = 0.0;
#pragma omp simd reduction(+:tmp) aligned(C,A,B:64)
				for (k = 0; k < N; k++) {
					tmp += A[N * i + k] * B[N * k + j];
				}
				test[N * i + j] = tmp;
			}
		}
	}

	for (i = 0; i < N; i++)
		for (j = 0; j < N; j++)
			if (equal(C[N * i + j], test[N * i + j]) == 1) {
				printf("\n wrong at (%d,%d) - %f %f", i, j, C[N * i + j], test[N * i + j]);
				return -1;
			}
	return 0;
}